
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <math.h>

//Perform Matrix Multiplication Utilizing the CPU Only
double cpuMult(int N, double *output) {
    // Number of bytes to allocate for N doubles
    size_t bytes = N * N * sizeof(double);
    // Allocate memory for arrays A, B, and C
    double *A = (double*)malloc(bytes);
    double *B = (double*)malloc(bytes);
    double *C = (double*)malloc(bytes);
    // Fill input arrays A and B
    for(int i = 0; i < N; i ++) {
        for(int j = 0; j < N; j ++) {
            A[i* N + j] = sin(i);
            B[i * N + j] = cos(j);
        }
    }
    //Start Clock
    clock_t t;
    t = clock();
    //Perform For Loop Multiplication of Matrices
    double sum;
    for (int i = 0; i < N; i++) {
        for(int j = 0; j < N; j ++) {
            sum = 0.f;
            for(int k = 0; k < N; k ++) {
                sum += A[i * N + k] * B[k * N + j];
            }
            C[i * N + j] = sum;
        }
    }
    //End Clock
    t = clock() - t;
    double time_taken = ((double)t)/CLOCKS_PER_SEC;

    //Copy C to Output Array
    memcpy(output, C, bytes);
    //Free Memory
    free(A);
    free(B);
    free(C);
    //Return the amount of time taken
    return time_taken;
}

#define TILE_DIM 64
//Kernel
__global__ void tile_matrix_multiply(double* A, double* B, double* C, int N) {
    double CValue = 0;
    int Row = blockIdx.y*TILE_DIM + threadIdx.y;
    int Col = blockIdx.x*TILE_DIM + threadIdx.x;
    __shared__ double As[TILE_DIM][TILE_DIM];
    __shared__ double Bs[TILE_DIM][TILE_DIM];

    for (int k = 0; k < (TILE_DIM + N - 1)/TILE_DIM; k++) {
         if (k*TILE_DIM + threadIdx.x < N && Row < N)
             As[threadIdx.y][threadIdx.x] = A[Row*N + k*TILE_DIM + threadIdx.x];
         else
             As[threadIdx.y][threadIdx.x] = 0.0;

         if (k*TILE_DIM + threadIdx.y < N && Col < N)
             Bs[threadIdx.y][threadIdx.x] = B[(k*TILE_DIM + threadIdx.y)*N + Col];
         else
             Bs[threadIdx.y][threadIdx.x] = 0.0;

         __syncthreads();

         for (int n = 0; n < TILE_DIM; ++n)
             CValue += As[threadIdx.y][n] * Bs[n][threadIdx.x];
         __syncthreads();
    }
    if (Row < N && Col < N)
        C[((blockIdx.y * blockDim.y + threadIdx.y)*N) +
           (blockIdx.x * blockDim.x)+ threadIdx.x] = CValue;
}

//Perform Matrix Multiplication Utilizing the GPU Only
double gpuMult(int N, double *output) {
    //Number of bytes to allocate for N doubles
    size_t bytes = N * N * sizeof(double);
    //Allocate memory for arrays A, B, and C
    double *A = (double*)malloc(bytes);
    double *B = (double*)malloc(bytes);
    double *C = (double*)malloc(bytes);
    //Fill input arrays A and B
    for(int i = 0; i < N; i ++) {
        for(int j = 0; j < N; j ++) {
            A[i* N + j] = sin(i);
            B[i * N + j] = cos(j);
        }
    }

    //Start Clock
    clock_t t;
    t = clock();
    // Allocate memory for arrays d_A, d_B, and d_C on device
    double *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);
    // Copy data from host arrays A and B to device arrays d_A and d_B
    hipMemcpy(d_A, A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, bytes, hipMemcpyHostToDevice);
    // Set execution configuration parameters
    dim3 thr_per_blk(TILE_DIM, TILE_DIM, 1);
    dim3 blk_in_grid((N + TILE_DIM - 1)/TILE_DIM, (N + TILE_DIM - 1)/TILE_DIM, 1);
    // Launch kernel
    tile_matrix_multiply<<< blk_in_grid, thr_per_blk >>>(d_A, d_B, d_C, N);
    // Copy data from device array d_C to host array C
    hipMemcpy(C, d_C, bytes, hipMemcpyDeviceToHost);
    //End Clock
    t = clock() - t;
    double time_taken = ((double)t)/CLOCKS_PER_SEC;

    // Free GPU memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    //Copy C to Output Array
    memcpy(output, C, bytes);
    //Free Memory
    free(A);
    free(B);
    free(C);
    //Return the amount of time taken
    return time_taken;
}

//Main program
int main(){
    int power = 20;
    int vectorLengths[power];
    //Fill vectorLengths with vector lengths we would like to test (Values are (1.5^x) where x is in the range of [0:power])
    for(int i = 1; i < power; i ++) {
        vectorLengths[i] = (pow(1.5,i) + 0.5);
    }
    int N;
    for(int x = 1; x < power; x ++) {
        N = vectorLengths[x];
        printf("Running Comparison for NxN Matrix with N = %d \n", N);

        // Number of bytes to allocate for N doubles
        size_t bytes = N * N * sizeof(double);
        // Allocate memory for output arrays
        double *CPU = (double*)malloc(bytes);
        double *GPU = (double*)malloc(bytes);

        //Execute CPU Multiplication
        double cpuTime = cpuMult(N, CPU);
        printf("CPU Vector Multiplication took %f seconds to execute \n", cpuTime);
        //Execute GPU Multiplication
        double gpuTime = gpuMult(N, GPU);
        printf("GPU Vector Multiplication took %f seconds to execute \n", gpuTime);

        double err = 0;
        //Compare the results of both executions; sum up total of discrepancies
        for (int ROW=0; ROW < N; ROW++) {
            for (int COL=0; COL < N; COL++) {
                err += CPU[ROW * N + COL] - GPU[ROW * N + COL];
            }
        }
        printf("Difference Between Results Matrices: %f \n", abs(err));
        printf("\n");

        free(CPU);
        free(GPU);
    }
    return 0;
}
